// Option: --use-dpcpp-extensions=intel_device_math
// Option: --use-experimental-features=bfloat16_math_functions

#include <hip/hip_runtime.h>
#include "hip/hip_bf16.h"
#include "hip/hip_fp16.h"

__global__ void test(__half2 h, __hip_bfloat162 b) {
  // Start
  h2sqrt(h /*__half2*/);
  h2sqrt(b /*__nv_bfloat162*/);
  // End
}
