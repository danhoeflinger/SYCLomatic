
#include <hip/hip_runtime.h>
#include "hipsolver.h"

void test(hipsolverHandle_t handle, hipsolverEigMode_t jobz,
          hipblasFillMode_t uplo, int n, const float *a, int lda, const float *w,
          hipsolverSyevjInfo_t params) {
  // Start
  int buffer_size;
  hipsolverDnSsyevj_bufferSize(
      handle /*cusolverDnHandle_t*/, jobz /*cusolverEigMode_t*/,
      uplo /*cublasFillMode_t*/, n /*int*/, a /*const float **/, lda /*int*/,
      w /*const float **/, &buffer_size /*int **/, params /*syevjInfo_t*/);
  // End
}
