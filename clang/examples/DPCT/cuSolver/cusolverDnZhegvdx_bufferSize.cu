
#include <hip/hip_runtime.h>
#include "hipsolver.h"

void test(hipsolverHandle_t handle, hipsolverEigType_t itype,
          hipsolverEigMode_t jobz, hipsolverEigRange_t range,
          hipblasFillMode_t uplo, int n, const hipDoubleComplex *a, int lda,
          const hipDoubleComplex *b, int ldb, double vl, double vu, int il,
          int iu, int *h_meig, const double *w) {
  // Start
  int buffer_size;
  hipsolverDnZhegvdx_bufferSize(
      handle /*cusolverDnHandle_t*/, itype /*cusolverEigType_t*/,
      jobz /*cusolverEigMode_t*/, range /*cusolverEigRange_t*/,
      uplo /*cublasFillMode_t*/, n /*int*/, a /*const cuDoubleComplex **/,
      lda /*int*/, b /*const cuDoubleComplex **/, ldb /*int*/, vl /*double*/,
      vu /*double*/, il /*int*/, iu /*int*/, h_meig /*int **/,
      w /*const double **/, &buffer_size /*int **/);
  // End
}
