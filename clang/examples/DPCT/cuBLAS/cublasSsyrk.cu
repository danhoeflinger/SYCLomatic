
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, hipblasFillMode_t upper_lower,
          hipblasOperation_t trans, int n, int k, const float *alpha,
          const float *a, int lda, const float *beta, float *c, int ldc) {
  // Start
  hipblasSsyrk(handle /*cublasHandle_t*/, upper_lower /*cublasFillMode_t*/,
              trans /*cublasOperation_t*/, n /*int*/, k /*int*/,
              alpha /*const float **/, a /*const float **/, lda /*int*/,
              beta /*const float **/, c /*float **/, ldc /*int*/);
  // End
}
