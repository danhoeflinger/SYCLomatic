
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, hipblasOperation_t trans, int m, int n, int kl,
          int ku, const double *alpha, const double *a, int lda,
          const double *x, int incx, const double *beta, double *y, int incy) {
  // Start
  hipblasDgbmv(handle /*cublasHandle_t*/, trans /*cublasOperation_t*/, m /*int*/,
              n /*int*/, kl /*int*/, ku /*int*/, alpha /*const double **/,
              a /*const double **/, lda /*int*/, x /*const double **/,
              incx /*int*/, beta /*const double **/, y /*double **/,
              incy /*int*/);
  // End
}
