
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, hipblasFillMode_t upper_lower,
          hipblasOperation_t trans, int64_t n, int64_t k, const float *alpha,
          const float *a, int64_t lda, const float *beta, float *c,
          int64_t ldc) {
  // Start
  hipblasSsyrk_64(handle /*cublasHandle_t*/, upper_lower /*cublasFillMode_t*/,
                 trans /*cublasOperation_t*/, n /*int64_t*/, k /*int64_t*/,
                 alpha /*const float **/, a /*const float **/, lda /*int64_t*/,
                 beta /*const float **/, c /*float **/, ldc /*int64_t*/);
  // End
}
