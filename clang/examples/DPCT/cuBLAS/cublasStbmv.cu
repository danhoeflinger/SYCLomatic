
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, hipblasFillMode_t upper_lower,
          hipblasOperation_t trans, hipblasDiagType_t unit_nonunit, int n, int k,
          const float *a, int lda, float *x, int incx) {
  // Start
  hipblasStbmv(handle /*cublasHandle_t*/, upper_lower /*cublasFillMode_t*/,
              trans /*cublasOperation_t*/, unit_nonunit /*cublasDiagType_t*/,
              n /*int*/, k /*int*/, a /*const float **/, lda /*int*/,
              x /*float **/, incx /*int*/);
  // End
}
