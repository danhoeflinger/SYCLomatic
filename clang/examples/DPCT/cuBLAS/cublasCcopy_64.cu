
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, int64_t n, const hipComplex *x, int64_t incx,
          hipComplex *y, int64_t incy) {
  // Start
  hipblasCcopy_64(handle /*cublasHandle_t*/, n /*int64_t*/,
                 x /*const cuComplex **/, incx /*int64_t*/, y /*cuComplex **/,
                 incy /*int64_t*/);
  // End
}
