
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, hipblasFillMode_t upper_lower,
          hipblasOperation_t trans, int n, int k, const hipDoubleComplex *alpha,
          const hipDoubleComplex *a, int lda, const hipDoubleComplex *b, int ldb,
          const hipDoubleComplex *beta, hipDoubleComplex *c, int ldc) {
  // Start
  hipblasZsyrkx(handle /*cublasHandle_t*/, upper_lower /*cublasFillMode_t*/,
               trans /*cublasOperation_t*/, n /*int*/, k /*int*/,
               alpha /*const cuDoubleComplex **/, a /*const cuDoubleComplex **/,
               lda /*int*/, b /*const cuDoubleComplex **/, ldb /*int*/,
               beta /*const cuDoubleComplex **/, c /*cuDoubleComplex **/,
               ldc /*int*/);
  // End
}
