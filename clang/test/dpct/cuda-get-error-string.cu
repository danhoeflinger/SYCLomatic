// RUN: dpct --format-range=none -out-root %T/cuda-get-error-string %s --cuda-include-path="%cuda-path/include" -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck %s --match-full-lines --input-file %T/cuda-get-error-string/cuda-get-error-string.dp.cpp
// RUN: %if build_lit %{icpx -c -fsycl %T/cuda-get-error-string/cuda-get-error-string.dp.cpp -o %T/cuda-get-error-string/cuda-get-error-string.dp.o %}

#include "hip/hip_runtime.h"

int printf(const char *format, ...);

// CHECK: /*
// CHECK-NEXT: DPCT1009:{{[0-9]+}}: SYCL reports errors using exceptions and does not use error codes. Please replace the "get_error_string_dummy(...)" with a real error-handling function.
// CHECK-NEXT: */
// CHECK-NEXT: #define PRINT_ERROR_STR(X) printf("%s\n", dpct::get_error_string_dummy(X))
#define PRINT_ERROR_STR(X) printf("%s\n", hipGetErrorString(X))

// CHECK:  /*
// CHECK-NEXT:  DPCT1009:{{[0-9]+}}: SYCL reports errors using exceptions and does not use error codes. Please replace the "get_error_string_dummy(...)" with a real error-handling function.
// CHECK-NEXT:  */
// CHECK-NEXT: #define PRINT_ERROR_STR2(X)\
// CHECK-NEXT:  printf("%s\n", dpct::get_error_string_dummy(X))
#define PRINT_ERROR_STR2(X)\
  printf("%s\n", hipGetErrorString(X))

// CHECK: /*
// CHECK-NEXT: DPCT1009:{{[0-9]+}}: SYCL reports errors using exceptions and does not use error codes. Please replace the "get_error_string_dummy(...)" with a real error-handling function.
// CHECK-NEXT: */
// CHECK-NEXT: #define PRINT_ERROR_STR3(X)\
// CHECK-NEXT:   printf("%s\
// CHECK-NEXT:          \n", dpct::get_error_string_dummy(X))
#define PRINT_ERROR_STR3(X)\
  printf("%s\
         \n", hipGetErrorString(X))

// CHECK: /*
// CHECK-NEXT: DPCT1009:{{[0-9]+}}: SYCL reports errors using exceptions and does not use error codes. Please replace the "get_error_string_dummy(...)" with a real error-handling function.
// CHECK-NEXT: */
// CHECK-NEXT: #define PRINT_ERROR_NAME(X) printf("%s\n", dpct::get_error_string_dummy(X))
#define PRINT_ERROR_NAME(X) printf("%s\n", hipGetErrorName(X))

// CHECK:   /*
// CHECK-NEXT:   DPCT1009:{{[0-9]+}}: SYCL reports errors using exceptions and does not use error codes. Please replace the "get_error_string_dummy(...)" with a real error-handling function.
// CHECK-NEXT:   */
// CHECK-NEXT: #define PRINT_ERROR_NAME2(X)\
// CHECK-NEXT:   printf("%s\n", dpct::get_error_string_dummy(X))
#define PRINT_ERROR_NAME2(X)\
  printf("%s\n", hipGetErrorName(X))

// CHECK: /*
// CHECK-NEXT: DPCT1009:{{[0-9]+}}: SYCL reports errors using exceptions and does not use error codes. Please replace the "get_error_string_dummy(...)" with a real error-handling function.
// CHECK-NEXT: */
// CHECK-NEXT: #define PRINT_ERROR_NAME3(X)\
// CHECK-NEXT:   printf("%s\
// CHECK-NEXT:          \n", dpct::get_error_string_dummy(X))
#define PRINT_ERROR_NAME3(X)\
  printf("%s\
         \n", hipGetErrorName(X))

// CHECK: /*
// CHECK-NEXT: DPCT1009:{{[0-9]+}}: SYCL reports errors using exceptions and does not use error codes. Please replace the "get_error_string_dummy(...)" with a real error-handling function.
// CHECK-NEXT: */
// CHECK-NEXT: #define PRINT_ERROR_STR_NAME(X)\
// CHECK-NEXT:   printf("%s\
// CHECK-NEXT:          %s\
// CHECK-NEXT:          \n", dpct::get_error_string_dummy(X),\
// CHECK-NEXT:          dpct::get_error_string_dummy(X))
#define PRINT_ERROR_STR_NAME(X)\
  printf("%s\
         %s\
         \n", hipGetErrorString(X),\
         hipGetErrorName(X))

const char *test_function() {
  // CHECK: /*
  // CHECK-NEXT: DPCT1010:{{[0-9]+}}: SYCL uses exceptions to report errors and does not use the error codes. The call was replaced with 0. You need to rewrite this code.
  // CHECK-NEXT: */
  PRINT_ERROR_STR(hipGetLastError());
  // CHECK: /*
  // CHECK-NEXT: DPCT1010:{{[0-9]+}}: SYCL uses exceptions to report errors and does not use the error codes. The call was replaced with 0. You need to rewrite this code.
  // CHECK-NEXT: */
  PRINT_ERROR_NAME(hipGetLastError());
  PRINT_ERROR_STR(hipSuccess);
  PRINT_ERROR_NAME(hipSuccess);

  // CHECK: /*
  // CHECK-NEXT: DPCT1010:{{[0-9]+}}: SYCL uses exceptions to report errors and does not use the error codes. The call was replaced with 0. You need to rewrite this code.
  // CHECK-NEXT: */
  PRINT_ERROR_STR2(hipGetLastError());
  // CHECK: /*
  // CHECK-NEXT: DPCT1010:{{[0-9]+}}: SYCL uses exceptions to report errors and does not use the error codes. The call was replaced with 0. You need to rewrite this code.
  // CHECK-NEXT: */
  PRINT_ERROR_NAME2(hipGetLastError());
  PRINT_ERROR_STR2(hipSuccess);
  PRINT_ERROR_NAME2(hipSuccess);

  // CHECK: /*
  // CHECK-NEXT: DPCT1010:{{[0-9]+}}: SYCL uses exceptions to report errors and does not use the error codes. The call was replaced with 0. You need to rewrite this code.
  // CHECK-NEXT: */
  PRINT_ERROR_STR3(hipGetLastError());
  // CHECK: /*
  // CHECK-NEXT: DPCT1010:{{[0-9]+}}: SYCL uses exceptions to report errors and does not use the error codes. The call was replaced with 0. You need to rewrite this code.
  // CHECK-NEXT: */
  PRINT_ERROR_NAME3(hipGetLastError());
  PRINT_ERROR_STR3(hipSuccess);
  PRINT_ERROR_NAME3(hipSuccess);

  // CHECK: /*
  // CHECK-NEXT: DPCT1010:{{[0-9]+}}: SYCL uses exceptions to report errors and does not use the error codes. The call was replaced with 0. You need to rewrite this code.
  // CHECK-NEXT: */
  PRINT_ERROR_STR_NAME(hipGetLastError());
  PRINT_ERROR_STR_NAME(hipSuccess);

//CHECK:/*
//CHECK-NEXT:DPCT1009:{{[0-9]+}}: SYCL reports errors using exceptions and does not use error codes. Please replace the "get_error_string_dummy(...)" with a real error-handling function.
//CHECK-NEXT:*/
//CHECK-NEXT:/*
//CHECK-NEXT:DPCT1010:{{[0-9]+}}: SYCL uses exceptions to report errors and does not use the error codes. The call was replaced with 0. You need to rewrite this code.
//CHECK-NEXT:*/
//CHECK-NEXT:  printf("%s\n", dpct::get_error_string_dummy(0));
  printf("%s\n", hipGetErrorString(hipGetLastError()));

//CHECK:  /*
//CHECK-NEXT:  DPCT1009:{{[0-9]+}}: SYCL reports errors using exceptions and does not use error codes. Please replace the "get_error_string_dummy(...)" with a real error-handling function.
//CHECK-NEXT:  */
//CHECK-NEXT:  printf("%s\n", dpct::get_error_string_dummy(0));
  printf("%s\n", hipGetErrorString(hipSuccess));

//CHECK:/*
//CHECK-NEXT:DPCT1009:{{[0-9]+}}: SYCL reports errors using exceptions and does not use error codes. Please replace the "get_error_string_dummy(...)" with a real error-handling function.
//CHECK-NEXT:*/
//CHECK-NEXT:printf("%s\n", dpct::get_error_string_dummy(0));
  printf("%s\n", hipGetErrorName(hipSuccess));
  hipError_t e;
  const char *err_s;

//CHECK:  /*
//CHECK-NEXT:  DPCT1009:{{[0-9]+}}: SYCL reports errors using exceptions and does not use error codes. Please replace the "get_error_string_dummy(...)" with a real error-handling function.
//CHECK-NEXT:  */
//CHECK-NEXT:  err_s = dpct::get_error_string_dummy(e);
  hipDrvGetErrorString(e, &err_s);

//CHECK:/*
//CHECK-NEXT:DPCT1009:{{[0-9]+}}: SYCL reports errors using exceptions and does not use error codes. Please replace the "get_error_string_dummy(...)" with a real error-handling function.
//CHECK-NEXT:*/
//CHECK-NEXT:  return dpct::get_error_string_dummy(0);
  return hipGetErrorName(hipSuccess);
}

//CHECK:void foo1(int err, const char *c) {
//CHECK-NEXT:  /*
//CHECK-NEXT:  DPCT1009:{{[0-9]+}}: SYCL reports errors using exceptions and does not use error codes. Please replace the "get_error_string_dummy(...)" with a real error-handling function.
//CHECK-NEXT:  */
//CHECK-NEXT:  c = dpct::get_error_string_dummy(err);
//CHECK-NEXT:  /*
//CHECK-NEXT:  DPCT1009:{{[0-9]+}}: SYCL reports errors using exceptions and does not use error codes. Please replace the "get_error_string_dummy(...)" with a real error-handling function.
//CHECK-NEXT:  */
//CHECK-NEXT:  c = dpct::get_error_string_dummy({{[0-9]+}});
//CHECK-NEXT:}
void foo1(hipError_t err, const char *c) {
  hipDrvGetErrorString(err, &c);
  hipDrvGetErrorString(hipErrorUnknown, &c);
}

//CHECK:void foo2(dpct::err0 err) {
//CHECK-NEXT:  /*
//CHECK-NEXT:  DPCT1009:{{[0-9]+}}: SYCL reports errors using exceptions and does not use error codes. Please replace the "get_error_string_dummy(...)" with a real error-handling function.
//CHECK-NEXT:  */
//CHECK-NEXT:  dpct::get_error_string_dummy(err);
//CHECK-NEXT:  /*
//CHECK-NEXT:  DPCT1009:{{[0-9]+}}: SYCL reports errors using exceptions and does not use error codes. Please replace the "get_error_string_dummy(...)" with a real error-handling function.
//CHECK-NEXT:  */
//CHECK-NEXT:  dpct::get_error_string_dummy({{[0-9]+}});
//CHECK-NEXT:}
void foo2(hipError_t err) {
  hipGetErrorString(err);
  hipGetErrorString(hipErrorInvalidValue);
}

void report_cuda_error(const char *stmt, const char *func, const char *file,
                       int line, const char *msg) {}

#define __CUDA_CHECK__(err, success, error_fn)                                 \
  do {                                                                         \
    auto err_ = (err);                                                         \
    if (err_ != (success)) {                                                   \
      report_cuda_error(#err, __func__, __FILE__, __LINE__, error_fn(err_));   \
    }                                                                          \
  } while (0)
// CHECK: #define CUDA_CHECK(err) __CUDA_CHECK__(err, 0, dpct::get_error_string_dummy)
#define CUDA_CHECK(err) __CUDA_CHECK__(err, hipSuccess, hipGetErrorString)

int main() {
  float *f;
  // CHECK: /*
  // CHECK-NEXT: DPCT1009:{{[0-9]+}}: SYCL reports errors using exceptions and does not use error codes. Please replace the "get_error_string_dummy(...)" with a real error-handling function.
  // CHECK-NEXT: */
  // CHECK-NEXT: CUDA_CHECK(DPCT_CHECK_ERROR(f = sycl::malloc_device<float>(1, q_ct1)));
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1009:{{[0-9]+}}: SYCL reports errors using exceptions and does not use error codes. Please replace the "get_error_string_dummy(...)" with a real error-handling function.
  // CHECK-NEXT: */
  // CHECK-NEXT: CUDA_CHECK(DPCT_CHECK_ERROR(dpct::dpct_free(f, q_ct1)));
  CUDA_CHECK(hipMalloc(&f, sizeof(float)));
  CUDA_CHECK(hipFree(f));
  return 0;
}
