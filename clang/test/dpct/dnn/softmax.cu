// RUN: dpct -in-root %S -out-root %T/softmax %S/softmax.cu --cuda-include-path="%cuda-path/include" -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/softmax/softmax.dp.cpp --match-full-lines %s

// CHECK: #include <sycl/sycl.hpp>
// CHECK-NEXT: #include <dpct/dpct.hpp>
// CHECK-NEXT: #include <dpct/dnnl_utils.hpp>
// CHECK-NEXT: #include <iostream>
// CHECK-NEXT: #include <vector>
#include <hip/hip_runtime.h>
#include <cudnn.h>
#include <iostream>
#include <vector>

// CHECK: template <dpct::library_data_t T>
// CHECK: struct dt_trait {
// CHECK:     typedef void type;
// CHECK: };
// CHECK: template <>
// CHECK: struct dt_trait<dpct::library_data_t::real_float> {
// CHECK:     typedef float type;
// CHECK: };
// CHECK: template <>
// CHECK: struct dt_trait<dpct::library_data_t::real_double> {
// CHECK:     typedef double type;
// CHECK: };
// CHECK: template <>
// CHECK: struct dt_trait<dpct::library_data_t::real_int32> {
// CHECK:     typedef int type;
// CHECK: };
// CHECK: template <>
// CHECK: struct dt_trait<dpct::library_data_t::real_half> {
// CHECK:     typedef float type;
// CHECK: };

template<cudnnDataType_t T>
struct dt_trait{
    typedef void type;
};
template<>
struct dt_trait<CUDNN_DATA_FLOAT>{
    typedef float type;
};
template<>
struct dt_trait<CUDNN_DATA_DOUBLE>{
    typedef double type;
};
template<>
struct dt_trait<CUDNN_DATA_INT32>{
    typedef int type;
};
template<>
struct dt_trait<CUDNN_DATA_HALF>{
    typedef float type;
};


template<cudnnDataType_t T, typename HT = typename dt_trait<T>::type>
void test1() {
    cudnnHandle_t handle;
    cudnnTensorDescriptor_t dataTensor, outTensor, diffdataTensor, diffoutTensor;

    cudnnCreate(&handle);

    hipStream_t stream1;
    hipStreamCreate(&stream1);
    cudnnSetStream(handle, stream1);

    cudnnCreateTensorDescriptor(&dataTensor);
    cudnnCreateTensorDescriptor(&outTensor);
    cudnnCreateTensorDescriptor(&diffdataTensor);
    cudnnCreateTensorDescriptor(&diffoutTensor);
    int n = 1, c = 2, h = 5, w = 5;
    int ele_num = n * c * h * w;

    //using HT = dt_trait<T>::type;

    cudnnSetTensor4dDescriptor(dataTensor, CUDNN_TENSOR_NCHW, T, n, c, h, w);
    cudnnSetTensor4dDescriptor(outTensor, CUDNN_TENSOR_NCHW, T, n, c, h, w);
    cudnnSetTensor4dDescriptor(diffdataTensor, CUDNN_TENSOR_NCHW, T, n, c, h, w);
    cudnnSetTensor4dDescriptor(diffoutTensor, CUDNN_TENSOR_NCHW, T, n, c, h, w);
    HT *data, *out, *diffdata, *diffout;
    std::vector<HT> host_data(ele_num);
    std::vector<HT> host_out(ele_num);
    std::vector<HT> host_diffdata(ele_num);
    std::vector<HT> host_diffout(ele_num);
    for(int i = 0; i < ele_num; i++) {
        host_data[i] = i * 0.1f;
        host_out[i] = i;
        host_diffdata[i] = i;
        host_diffout[i] = 1.f;
    }

    hipMalloc(&data, ele_num * sizeof(HT));
    hipMalloc(&out, ele_num * sizeof(HT));
    hipMalloc(&diffdata, ele_num * sizeof(HT));
    hipMalloc(&diffout, ele_num * sizeof(HT));

    hipMemcpy(data, host_data.data(), ele_num * sizeof(HT), hipMemcpyHostToDevice);
    hipMemcpy(out, host_out.data(), ele_num * sizeof(HT), hipMemcpyHostToDevice);
    hipMemcpy(diffdata, host_diffdata.data(), ele_num * sizeof(HT), hipMemcpyHostToDevice);
    hipMemcpy(diffout, host_diffout.data(), ele_num * sizeof(HT), hipMemcpyHostToDevice);

    float alpha = 1.5f, beta = 0.f;
    // CHECK: handle.async_softmax_forward(dpct::dnnl::softmax_algorithm::normal, dpct::dnnl::softmax_mode::channel, alpha, dataTensor, data, beta, outTensor, out);
    // CHECK: dpct::get_in_order_queue().memcpy(host_out.data(), out, ele_num * sizeof(HT)).wait();
    // CHECK: alpha = 2.f, beta = 0.f;
    // CHECK: dpct::get_current_device().queues_wait_and_throw();
    // CHECK: auto s = DPCT_CHECK_ERROR(handle.async_softmax_backward(dpct::dnnl::softmax_algorithm::normal, dpct::dnnl::softmax_mode::channel, alpha, outTensor, out, diffoutTensor, diffout, beta, diffdataTensor, diffdata));

    cudnnSoftmaxForward(handle, CUDNN_SOFTMAX_ACCURATE, CUDNN_SOFTMAX_MODE_CHANNEL, &alpha, dataTensor, data, &beta, outTensor, out);
    hipMemcpy(host_out.data(), out, ele_num * sizeof(HT), hipMemcpyDeviceToHost);
    alpha = 2.f, beta = 0.f;
    hipDeviceSynchronize();
    auto s = cudnnSoftmaxBackward(handle, CUDNN_SOFTMAX_ACCURATE, CUDNN_SOFTMAX_MODE_CHANNEL, &alpha, outTensor, out, diffoutTensor, diffout, &beta, diffdataTensor, diffdata);
    hipDeviceSynchronize();

    hipMemcpy(host_diffdata.data(), diffdata, ele_num * sizeof(HT), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    cudnnDestroy(handle);
    hipFree(data);
    hipFree(out);
    hipFree(diffdata);
    hipFree(diffout);
}

int main() {
    int nDevices;
    hipGetDeviceCount(&nDevices);
    hipSetDevice(1);
    
    test1<CUDNN_DATA_FLOAT>();

    return 0;
}