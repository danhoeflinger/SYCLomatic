// RUN: cat %s > %T/error-handling.cu
// RUN: cd %T
// RUN: dpct --usm-level=none -out-root %T/error-handling error-handling.cu --cuda-include-path="%cuda-path/include" -- -w -x cuda --cuda-host-only -std=c++11
// RUN: FileCheck error-handling.cu --match-full-lines --input-file %T/error-handling/error-handling.dp.cpp
// RUN: %if build_lit %{icpx -c -fsycl -DNO_BUILD_TEST  %T/error-handling/error-handling.dp.cpp -o %T/error-handling/error-handling.dp.o %}

#include "hip/hip_runtime.h"
#include <hipblas.h>
#include <stdexcept>
#include <vector>
int printf(const char *s, ...);
int fprintf(int, const char *s, ...);

// CHECK:void test_simple_ifs() {
// CHECK-NEXT:  dpct::err0 err;
// CHECK-NEXT:}
void test_simple_ifs() {
  hipError_t err;
  if (err != hipSuccess) {
  }
  if (err) {
  }
  if (err != 0) {
  }
  if (0 != err) {
  }
  if (hipSuccess != err) {
  }
  if (err != hipSuccess) {
  }
}

// CHECK:void test_simple_ifs_const() {
// CHECK-NEXT:  const dpct::err0 err = 0;
// CHECK-NEXT:}
void test_simple_ifs_const() {
  const hipError_t err = hipSuccess;
  if (err != hipSuccess) {
  }
  if (err) {
  }
  if (err != 0) {
  }
  if (0 != err) {
  }
  if (hipSuccess != err) {
  }
  if (err != hipSuccess) {
  }
}

// CHECK:void test_typedef() {
// CHECK-NEXT:  typedef dpct::err0 someError_t;
// CHECK-NEXT:  someError_t err;
// CHECK-NEXT:}
void test_typedef() {
  typedef hipError_t someError_t;
  someError_t err;
  if (err != hipSuccess) {
  }
  if (0 != err) {
  }
}

// CHECK:void test_no_braces() {
// CHECK-NEXT:  dpct::err0 err;
// CHECK-NEXT:}
void test_no_braces() {
  hipError_t err;
  if (err != hipSuccess)
    printf("error!\n");
}

// CHECK:void test_unrelated_then() {
// CHECK-NEXT:  dpct::err0 err;
// CHECK-NEXT:  int i = 0;
// CHECK-NEXT:/*
// CHECK-NEXT:DPCT1000:{{[0-9]+}}: Error handling if-stmt was detected but could not be rewritten.
// CHECK-NEXT:*/
// CHECK-NEXT:   if (err != 0) {
// CHECK-NEXT:/*
// CHECK-NEXT:DPCT1001:{{[0-9]+}}: The statement could not be removed.
// CHECK-NEXT:*/
// CHECK-NEXT:    ++i;
// CHECK-NEXT:  }
// CHECK-NEXT:}

void test_unrelated_then() {
  hipError_t err;
  int i = 0;
  if (err != hipSuccess) {
    ++i;
  }
}

// CHECK:void test_CUDA_SUCCESS() {
// CHECK-NEXT:  dpct::err0 err;
// CHECK-NEXT:}
void test_CUDA_SUCCESS() {
  hipError_t err;
  if (err != hipSuccess) {
    printf("error!\n");
  }
}

// CHECK:void test_CUDA_SUCCESS_empty() {
// CHECK-NEXT:  dpct::err0 err;
// CHECK-NEXT:}
void test_CUDA_SUCCESS_empty() {
  hipError_t err;
  if (err != hipSuccess) {
  }
}

// CHECK:void test_CUDA_SUCCESS_CUresult() {
// CHECK-NEXT:  int err;
// CHECK-NEXT:}
void test_CUDA_SUCCESS_CUresult() {
  hipError_t err;
  if (err != hipSuccess) {
    printf("error!\n");
  }
}

// CHECK:void test_CUDA_SUCCESS_empty_CUresult() {
// CHECK-NEXT:  int err;
// CHECK-NEXT:}
void test_CUDA_SUCCESS_empty_CUresult() {
  hipError_t err;
  if (err != hipSuccess) {
  }
}

// CHECK:void test_other_enum() {
// CHECK-NEXT:  dpct::err0 err;
// CHECK-NEXT:  if (err != {{[0-9]+}}) {
// CHECK-NEXT:    printf("error!\n");
// CHECK-NEXT:  }
// CHECK-NEXT:}
void test_other_enum() {
  hipError_t err;
  if (err != hipErrorLaunchFailure) {
    printf("error!\n");
  }
}

#ifndef NO_BUILD_TEST
// CHECK:void test_assignment() try {
// CHECK-NEXT:  dpct::err0 err;
// CHECK-NEXT:  if (err = DPCT_CHECK_ERROR(* 0 = dpct::dpct_malloc(0))) {
// CHECK-NEXT:    printf("error!\n");
// CHECK-NEXT:  }
// CHECK-NEXT:}
// CHECK-NEXT: catch (sycl::exception const &exc) {
// CHECK-NEXT:   std::cerr << exc.what() << "Exception caught at file:" << __FILE__
// CHECK-NEXT:             << ", line:" << __LINE__ << std::endl;
// CHECK-NEXT:   std::exit(1);
// CHECK-NEXT: }
void test_assignment() {
  hipError_t err;
  if (err = hipMalloc(0, 0)) {
    printf("error!\n");
  }
}
#endif

// CHECK:void test_1(dpct::err0 err, int arg) {
// CHECK-NEXT:  if (err == 0 && arg) {
// CHECK-NEXT:  }
// CHECK-NEXT:}
void test_1(hipError_t err, int arg) {
  if (err == hipSuccess && arg) {
  }
}

// CHECK:void test_12(dpct::err0 err, int arg) {
// CHECK-NEXT:  if (err) {
// CHECK-NEXT:  } else {
// CHECK-NEXT:  {{ +}}
// CHECK-NEXT:  }
// CHECK-NEXT:}
void test_12(hipError_t err, int arg) {
  if (err) {
  } else {
    
  }
}

// CHECK:void test_13(dpct::err0 err, int arg) {
// CHECK-NEXT:}
void test_13(hipError_t err, int arg) {
  if (err) {
    printf("error!\n");
  }
}

// CHECK:void test_14(dpct::err0 err, int arg) {
// CHECK-NEXT:  if (arg == 1) {
// CHECK-NEXT:    return;
// CHECK-NEXT:  }
// CHECK-NEXT  if (arg != 0) {
// CHECK-NEXT    return;
// CHECK-NEXT  }
// CHECK-NEXT  if (arg) {
// CHECK-NEXT    return;
// CHECK-NEXT  }
// CHECK-NEXT}
void test_14(hipError_t err, int arg) {
  if (arg == 1) {
    return;
  }
  if (arg != 0) {
    return;
  }
  if (arg) {
    return;
  }
}

#ifndef NO_BUILD_TEST
// CHECK:void test_15(dpct::err0 err, int arg) try {
// CHECK-NEXT:  if (DPCT_CHECK_ERROR(* 0 = dpct::dpct_malloc(0))) {
// CHECK-NEXT:  }
// CHECK-NEXT:}
// CHECK-NEXT: catch (sycl::exception const &exc) {
// CHECK-NEXT:   std::cerr << exc.what() << "Exception caught at file:" << __FILE__
// CHECK-NEXT:             << ", line:" << __LINE__ << std::endl;
// CHECK-NEXT:   std::exit(1);
// CHECK-NEXT: }
void test_15(hipError_t err, int arg) {
  if (hipMalloc(0, 0)) {
  }
}
#endif

// CHECK:void test_16(dpct::err0 err, int arg) {
// CHECK-NEXT:  if (err) {
// CHECK-NEXT:    printf("error!\n");
// CHECK-NEXT:    exit(1);
// CHECK-NEXT:  } else {
// CHECK-NEXT:{{ +}}
// CHECK-NEXT:  }
// CHECK-NEXT:}
void test_16(hipError_t err, int arg) {
  if (err) {
    printf("error!\n");
    exit(1);
  } else {
    
  }
}

#ifndef NO_BUILD_TEST
// CHECK:void test_17(dpct::err0 err, int arg)  try {
// CHECK-NEXT:  if (!DPCT_CHECK_ERROR(* 0 = dpct::dpct_malloc(0))) {
// CHECK-NEXT:  } else {
// CHECK-NEXT:    printf("error!\n");
// CHECK-NEXT:    exit(1);
// CHECK-NEXT:  }
// CHECK-NEXT:}
// CHECK-NEXT: catch (sycl::exception const &exc) {
// CHECK-NEXT:   std::cerr << exc.what() << "Exception caught at file:" << __FILE__
// CHECK-NEXT:             << ", line:" << __LINE__ << std::endl;
// CHECK-NEXT:   std::exit(1);
// CHECK-NEXT: }
void test_17(hipError_t err, int arg) {
  if (!hipMalloc(0, 0)) {
  } else {
    printf("error!\n");
    exit(1);
  }
}
#endif

// CHECK:void test_18(dpct::err0 err, int arg) {
// CHECK-NEXT:  if (err)
// CHECK-NEXT:    printf("error!\n");
// CHECK-NEXT:  else
// CHECK-NEXT:    printf("success!\n");
// CHECK-NEXT:}
void test_18(hipError_t err, int arg) {
  if (err)
    printf("error!\n");
  else
    printf("success!\n");
}

// CHECK:void test_19(dpct::err0 err, int arg) {
// CHECK-NEXT:  if (err && arg) {
// CHECK-NEXT:  } else {
// CHECK-NEXT:  }
// CHECK-NEXT:}
void test_19(hipError_t err, int arg) {
  if (err && arg) {
  } else {
  }
}

// CHECK:void test_compare_to_3(dpct::err0 err, int arg) {
// CHECK-NEXT:  if (err != 3) {
// CHECK-NEXT:  }
// CHECK-NEXT:}
void test_compare_to_3(hipError_t err, int arg) {
  if (err != 3) {
  }
}

// CHECK:void test_21(const dpct::err0 &err, int arg) {
// CHECK-NEXT:}
void test_21(const hipError_t &err, int arg) {
  if (err != 0) {
  }
}

// CHECK:void test_no_side_effects(dpct::err0 err, int arg) {
// CHECK-NEXT:}
void test_no_side_effects(hipError_t err, int arg) {
  if (err)
    printf("efef");
  if (err)
    fprintf(0, "efef");
  if (err)
    exit(1);
  if (err != hipSuccess) {
    printf("error!\n");
    exit(1);
  }
}

// CHECK:void test_side_effects(dpct::err0 err, int arg, int x, int y, int z) {
// CHECK:/*
// CHECK-NEXT:DPCT1000:{{[0-9]+}}: Error handling if-stmt was detected but could not be rewritten.
// CHECK-NEXT:*/
// CHECK-NEXT:  if (err)
// CHECK-NEXT:/*
// CHECK-NEXT:DPCT1001:{{[0-9]+}}: The statement could not be removed.
// CHECK-NEXT:*/
// CHECK-NEXT:    malloc(0x100);
// CHECK-NEXT:/*
// CHECK-NEXT:DPCT1000:{{[0-9]+}}: Error handling if-stmt was detected but could not be rewritten.
// CHECK-NEXT:*/
// CHECK-NEXT:  if (err != 0) {
// CHECK-NEXT:/*
// CHECK-NEXT:DPCT1001:{{[0-9]+}}: The statement could not be removed.
// CHECK-NEXT:*/
// CHECK-NEXT:    malloc(0x100);
// CHECK-NEXT:    printf("error!\n");
// CHECK-NEXT:    exit(1);
// CHECK-NEXT:  }
// CHECK-NEXT:/*
// CHECK-NEXT:DPCT1000:{{[0-9]+}}: Error handling if-stmt was detected but could not be rewritten.
// CHECK-NEXT:*/
// CHECK-NEXT:  if (err)
// CHECK-NEXT:/*
// CHECK-NEXT:DPCT1001:{{[0-9]+}}: The statement could not be removed.
// CHECK-NEXT:*/
// CHECK-NEXT:    x = printf("fmt string");
// CHECK-NEXT:}

void test_side_effects(hipError_t err, int arg, int x, int y, int z) {
  if (err)
    printf("efef %i", malloc(0x100));
  if (err)
    malloc(0x100);
  if (err != hipSuccess) {
    malloc(0x100);
    printf("error!\n");
    exit(1);
  }
  if (err)
    x = printf("fmt string");
  if (err)
    printf("fmt string %d", y + z);
}

// CHECK:void specialize_ifs() {
// CHECK-NEXT:  dpct::err0 err;
// CHECK-NEXT:}
void specialize_ifs() {
  hipError_t err;
  if (err == hipErrorAssert) {
    printf("efef");
  }
  if (err == 255) {
  }
  if (err == 1) {
  }
  if (666 == err) {
  }
  if (hipErrorAssert == err) {
  }
}

// CHECK:void specialize_ifs_negative() {
// CHECK-NEXT:  dpct::err0 err;
// CHECK-NEXT:  if (err == 0) {
// CHECK-NEXT:    printf("efef");
// CHECK-NEXT:  }
// CHECK-NEXT:/*
// CHECK-NEXT:DPCT1002:{{[0-9]+}}: Special case error handling if-stmt was detected. You may need to
// CHECK-NEXT:rewrite this code.
// CHECK-NEXT:*/
// CHECK-NEXT:  if (err == {{[0-9]+}}) {
// CHECK-NEXT:    printf("efef");
// CHECK-NEXT:/*
// CHECK-NEXT:DPCT1001:{{[0-9]+}}: The statement could not be removed.
// CHECK-NEXT:*/
// CHECK-NEXT:    malloc(0x100);
// CHECK-NEXT:  }
// CHECK-NEXT:/*
// CHECK-NEXT:DPCT1002:{{[0-9]+}}: Special case error handling if-stmt was detected. You may need to
// CHECK-NEXT:rewrite this code.
// CHECK-NEXT:*/
// CHECK-NEXT:  if (err == 255) {
// CHECK-NEXT:/*
// CHECK-NEXT:DPCT1001:{{[0-9]+}}: The statement could not be removed.
// CHECK-NEXT:*/
// CHECK-NEXT:    malloc(0x100);
// CHECK-NEXT:  }
// CHECK-NEXT:/*
// CHECK-NEXT:DPCT1002:{{[0-9]+}}: Special case error handling if-stmt was detected. You may need to
// CHECK-NEXT:rewrite this code.
// CHECK-NEXT:*/
// CHECK-NEXT:  if (err == 1) {
// CHECK-NEXT:/*
// CHECK-NEXT:DPCT1001:{{[0-9]+}}: The statement could not be removed.
// CHECK-NEXT:*/
// CHECK-NEXT:    malloc(0x100);
// CHECK-NEXT:  }
// CHECK-NEXT:/*
// CHECK-NEXT:DPCT1002:{{[0-9]+}}: Special case error handling if-stmt was detected. You may need to
// CHECK-NEXT:rewrite this code.
// CHECK-NEXT:*/
// CHECK-NEXT:  if (666 == err) {
// CHECK-NEXT:/*
// CHECK-NEXT:DPCT1001:{{[0-9]+}}: The statement could not be removed.
// CHECK-NEXT:*/
// CHECK-NEXT:    malloc(0x100);
// CHECK-NEXT:  }
// CHECK-NEXT:/*
// CHECK-NEXT:DPCT1002:{{[0-9]+}}: Special case error handling if-stmt was detected. You may need to
// CHECK-NEXT:rewrite this code.
// CHECK-NEXT:*/
// CHECK-NEXT:  if ({{[0-9]+}} == err) {
// CHECK-NEXT:/*
// CHECK-NEXT:DPCT1001:{{[0-9]+}}: The statement could not be removed.
// CHECK-NEXT:*/
// CHECK-NEXT:    malloc(0x100);
// CHECK-NEXT:  }
// CHECK-NEXT:}
void specialize_ifs_negative() {
  hipError_t err;
  if (err == hipSuccess) {
    printf("efef");
  }
  if (err == hipErrorAssert) {
    printf("efef");
    malloc(0x100);
  }
  if (err == 255) {
    malloc(0x100);
  }
  if (err == 1) {
    malloc(0x100);
  }
  if (666 == err) {
    malloc(0x100);
  }
  if (hipErrorAssert == err) {
    malloc(0x100);
  }
}

#ifndef NO_BUILD_TEST
// CHECK: void foo1() try {
// CHECK-NEXT:   if (DPCT_CHECK_ERROR(* 0 = dpct::dpct_malloc(0))) {
// CHECK-NEXT:     printf("efef");
// CHECK-NEXT:   }
// CHECK-NEXT: }
// CHECK-NEXT: catch (sycl::exception const &exc) {
// CHECK-NEXT:   std::cerr << exc.what() << "Exception caught at file:" << __FILE__
// CHECK-NEXT:             << ", line:" << __LINE__ << std::endl;
// CHECK-NEXT:   std::exit(1);
// CHECK-NEXT: }
void foo1() {
  if (hipMalloc(0, 0)) {
    printf("efef");
  }
}

// CHECK: void foo2() try {
// CHECK-NEXT:   for (; DPCT_CHECK_ERROR(* 0 = dpct::dpct_malloc(0));) {
// CHECK-NEXT:     printf("efef");
// CHECK-NEXT:   }
// CHECK-NEXT: }
// CHECK-NEXT: catch (sycl::exception const &exc) {
// CHECK-NEXT:   std::cerr << exc.what() << "Exception caught at file:" << __FILE__
// CHECK-NEXT:             << ", line:" << __LINE__ << std::endl;
// CHECK-NEXT:   std::exit(1);
// CHECK-NEXT: }
void foo2() {
  for (; hipMalloc(0, 0);) {
    printf("efef");
  }
}

// CHECK: void foo3() try {
// CHECK-NEXT:   do{
// CHECK-NEXT:     printf("efef");
// CHECK-NEXT:   } while (DPCT_CHECK_ERROR(* 0 = dpct::dpct_malloc(0)));
// CHECK-NEXT: }
// CHECK-NEXT: catch (sycl::exception const &exc) {
// CHECK-NEXT:   std::cerr << exc.what() << "Exception caught at file:" << __FILE__
// CHECK-NEXT:             << ", line:" << __LINE__ << std::endl;
// CHECK-NEXT:   std::exit(1);
// CHECK-NEXT: }
void foo3() {
  do{
    printf("efef");
  } while (hipMalloc(0, 0));
}

// CHECK: void foo4() {
// CHECK-NEXT:   int a;
// CHECK-NEXT:   a = DPCT_CHECK_ERROR(* 0 = dpct::dpct_malloc(0));
// CHECK-NEXT: }
void foo4() {
  int a;
  a = hipMalloc(0, 0);
}

// CHECK: void foo5() {
// CHECK-NEXT:   *0 = dpct::dpct_malloc(0);
// CHECK-NEXT:   int a = printf("a");
// CHECK-NEXT:   if(printf("a")) {}
// CHECK-NEXT:   dpct::event_ptr start;
// CHECK:   int b = DPCT_CHECK_ERROR(start = new sycl::event());
// CHECK-NEXT: }
void foo5() {
  hipMalloc(0, 0);
  int a = printf("a");
  if(printf("a")) {}
  hipEvent_t start;
  int b = hipEventCreate(&start);
}

// CHECK: void foo6() try {
// CHECK-NEXT:   int a = DPCT_CHECK_ERROR(* 0 = dpct::dpct_malloc(0));
// CHECK-NEXT:   if(a) printf("a");
// CHECK-NEXT: }
// CHECK-NEXT: catch (sycl::exception const &exc) {
// CHECK-NEXT: std::cerr << exc.what() << "Exception caught at file:" << __FILE__
// CHECK-NEXT:             << ", line:" << __LINE__ << std::endl;
// CHECK-NEXT: std::exit(1);
// CHECK-NEXT: }
void foo6() {
  int a = hipMalloc(0, 0);
  if(a) printf("a");
}

// CHECK: void foo7() try {
// CHECK-NEXT:   int a;
// CHECK-NEXT:   a = DPCT_CHECK_ERROR(* 0 = dpct::dpct_malloc(0));
// CHECK-NEXT:   if(a) printf("a");
// CHECK-NEXT: }
// CHECK-NEXT: catch (sycl::exception const &exc) {
// CHECK-NEXT: std::cerr << exc.what() << "Exception caught at file:" << __FILE__
// CHECK-NEXT:             << ", line:" << __LINE__ << std::endl;
// CHECK-NEXT: std::exit(1);
// CHECK-NEXT: }
void foo7() {
  int a;
  a = hipMalloc(0, 0);
  if(a) printf("a");
}

// CHECK: int foo8() try {
// CHECK-NEXT:   return DPCT_CHECK_ERROR(*0 = dpct::dpct_malloc(0));
// CHECK-NEXT: }
// CHECK-NEXT: catch (sycl::exception const &exc) {
// CHECK-NEXT: std::cerr << exc.what() << "Exception caught at file:" << __FILE__
//  CHECK-NEXT:             << ", line:" << __LINE__ << std::endl;
// CHECK-NEXT: std::exit(1);
// CHECK-NEXT: }
int foo8() {
  return hipMalloc(0, 0);
}

// CHECK: int foo9() {
// CHECK-NEXT: while (true) {
// CHECK-NEXT:   *0 = dpct::dpct_malloc(0);
// CHECK-NEXT: }
// CHECK-NEXT: }
int foo9() {
  while (true) {
    hipMalloc(0, 0);
  }
}

// CHECK: void foo10() try {
// CHECK-NEXT:   switch (DPCT_CHECK_ERROR(* 0 = dpct::dpct_malloc(0))) {
// CHECK-NEXT:     case 0:
// CHECK-NEXT:       break;
// CHECK-NEXT:     case 1:
// CHECK-NEXT:       break;
// CHECK-NEXT:     default:
// CHECK-NEXT:     ;
// CHECK-NEXT:   }
// CHECK-NEXT: }
// CHECK-NEXT: catch (sycl::exception const &exc) {
// CHECK-NEXT: std::cerr << exc.what() << "Exception caught at file:" << __FILE__
//  CHECK-NEXT:             << ", line:" << __LINE__ << std::endl;
// CHECK-NEXT: std::exit(1);
// CHECK-NEXT: }
void foo10() {
  switch (hipMalloc(0, 0)){
  case 0:
    break;
  case 1:
    break;
  default:
  ;
  }
}

// CHECK: class ClassA {
// CHECK-NEXT:   public:
// CHECK-NEXT:   std::vector<int> V;
// CHECK-NEXT:   ClassA() : V() {}
// CHECK-NEXT:   ClassA(int b) try {
// CHECK-NEXT:     int a = DPCT_CHECK_ERROR(* 0 = dpct::dpct_malloc(0));
// CHECK-NEXT:     if(a) { printf("a"); }
// CHECK-NEXT:   }
// CHECK-NEXT:   catch (sycl::exception const &exc) {
// CHECK-NEXT:     std::cerr << exc.what() << "Exception caught at file:" << __FILE__
// CHECK-NEXT:               << ", line:" << __LINE__ << std::endl;
// CHECK-NEXT:     std::exit(1);
// CHECK-NEXT:   }
// CHECK-NEXT: };
class ClassA {
public:
  std::vector<int> V;
  ClassA() : V() {}
  ClassA(int b) {
    int a = hipMalloc(0, 0);
    if(a) { printf("a"); }
  }
};
#endif

// CHECK: int foo11() {
// CHECK-NEXT:   int *a;
// CHECK-NEXT:   return [&]() {
/// FIXME: cudaMalloc is matched here, should be fixed.
// NOT-CHECK-NEXT:     try {
// CHECK-NEXT:       return DPCT_CHECK_ERROR(a = (int *)dpct::dpct_malloc(0));
// NOT-CHECK-NEXT:     }
// NOT-CHECK-NEXT:     catch (sycl::exception const &exc) {
// NOT-CHECK-NEXT:       std::cerr << exc.what() << "Exception caught at file:" << __FILE__
// NOT-CHECK-NEXT:                 << ", line:" << __LINE__ << std::endl;
// NOT-CHECK-NEXT:       std::exit(1);
// NOT-CHECK-NEXT:     }
// CHECK-NEXT:   }();
// CHECK-NEXT: }
int foo11() {
  int *a;
  return [&]() {
    return hipMalloc((void **)&a, 0);
  }();
}

// CHECK: void foo12() try {
// CHECK-NEXT:   size_t size = 1234567 * sizeof(float);
// CHECK-NEXT:   float *h_A = (float *)malloc(size);
// CHECK-NEXT:   float *d_A = NULL;
// CHECK-NEXT:   /*
// CHECK-NEXT:   DPCT1124:{{[0-9]+}}: cudaMemcpyAsync is migrated to asynchronous memcpy API. While the
// CHECK-NEXT:   origin API might be synchronous, it depends on the type of operand memory, so
// CHECK-NEXT:   you may need to call wait() on event return by memcpy API to ensure
// CHECK-NEXT:   synchronization behavior.
// CHECK-NEXT:   */
// CHECK-NEXT:   while (DPCT_CHECK_ERROR(
// CHECK-NEXT:       dpct::async_dpct_memcpy(d_A, h_A, size, dpct::host_to_device))) {
// CHECK-NEXT:     printf("efef");
// CHECK-NEXT:   }
// CHECK-NEXT: }
// CHECK-NEXT: catch (sycl::exception const &exc) {
// CHECK-NEXT:   std::cerr << exc.what() << "Exception caught at file:" << __FILE__
// CHECK-NEXT:             << ", line:" << __LINE__ << std::endl;
// CHECK-NEXT:   std::exit(1);
// CHECK-NEXT: }
void foo12() {
  size_t size = 1234567 * sizeof(float);
  float *h_A = (float *)malloc(size);
  float *d_A = NULL;
  while (hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice)) {
    printf("efef");
  }
}

// CHECK: void foo13() {
// CHECK-NEXT:   int res;
// CHECK-NEXT:   {
// CHECK-NEXT:   auto ct_0_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(0);
// CHECK-NEXT:   sycl::buffer<int64_t> res_temp_buf_ct{{[0-9]+}}(sycl::range<1>(1));
// CHECK-NEXT:   oneapi::mkl::blas::column_major::iamax(
// CHECK-NEXT:       dpct::blas::descriptor::get_saved_queue(), 10, ct_0_buf_ct{{[0-9]+}}, 0,
// CHECK-NEXT:       res_temp_buf_ct{{[0-9]+}});
// CHECK-NEXT:   res = res_temp_buf_ct{{[0-9]+}}.get_access<sycl::access_mode::read>()[0];
// CHECK-NEXT:   }
// CHECK-NEXT: }
void foo13() {
  int res = cublasIsamax(10, 0, 0);
}

// CHECK: __dpct_inline__ dpct::err0 foo14(dpct::err0 error, const char *filename,
// CHECK-NEXT:                                 int line) {
// CHECK-NEXT:  dpct::err0 error_1 = 0;
// CHECK-NEXT:  return error_1;
// CHECK-NEXT: }
__host__ __device__ __forceinline__ hipError_t foo14(hipError_t error,
                                                      const char *filename,
                                                      int line) {
  hipError_t error_1 = hipSuccess;
  return error_1;
}

// CHECK: __dpct_inline__ dpct::err0 *foo15(dpct::err0 error, const char *filename,
// CHECK-NEXT:                              int line) {
// CHECK-NEXT:  return &error;
// CHECK-NEXT: }
__host__ __device__ __forceinline__ hipError_t *foo15(hipError_t error,
                                                       const char *filename,
                                                       int line) {
  return &error;
}

// CHECK: int foo16() {
// CHECK-NEXT:   /*
// CHECK-NEXT:   DPCT1010:{{[0-9]+}}: SYCL uses exceptions to report errors and does not use the error
// CHECK-NEXT:   codes. The call was replaced with 0. You need to rewrite this code.
// CHECK-NEXT:   */
// CHECK-NEXT:   return 0;
// CHECK-NEXT: }
int foo16() {
  return hipGetLastError();
}
